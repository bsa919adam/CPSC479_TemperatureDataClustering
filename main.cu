
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
//use high as x value low as y for clustering
typedef struct day{
	int month;
	int date;
	int year;
	double high;
	double low;
	int cluster;
}day;

typedef struct center{
	double x;
	double y;
}center;


__global__ void setCenters(day* data, center* centers, int k, int numDays) {
	__shared__ int nums[3];

	int * avgx =nums;
	int * avgy =&nums[1];
	int * n=&nums[2];
	*avgx=0;
	*avgy=0;
	*n=0;
	int i =0;
	int index;	
	while((index=threadIdx.x + blockDim.x*i) < numDays){
		if(data[index].cluster==blockIdx.x){
			atomicAdd(n, 1);
			atomicAdd(avgx, (int)data[index].high);
			atomicAdd(avgy, (int)data[index].low);
			
		}
		
		i++;
	}
	__syncthreads();
	if(threadIdx.x ==0){
		centers[blockIdx.x].x=(double)(*avgx)/(double)(*n);
		centers[blockIdx.x].y=(double)(*avgy)/(double)(*n);
	}
	
}

__global__ void cluster(day* data, center* centers, int k, int numDays, int * s){
	int numT=gridDim.x*blockDim.x;
	int i=0;
	int index;
	int cluster;
	*s=0;
	while((index=threadIdx.x +blockIdx.x * blockDim.x+ numT*i) < numDays){
		double min=1000;
		for( int j=0; j< k; j++){
			double x=data[index].high-centers[j].x;
			x=x*x;
			double y=data[index].low-centers[j].y;
			y=y*y;
			double dist=sqrt(x+y);
			if(dist< min){
				min=dist;
				cluster=j;
				
			}		
		}
		if(data[index].cluster!=cluster){
			atomicAdd(s, 1);
			data[index].cluster=cluster;
		}
		i++;
	} 
}

int main(int  argc, char *argv[]) {
    printf("begin checks\n");
    if(argc < 3){ //checcks for proper number of Args
		printf("Missing Arguments");
		return 1;
    }
    int k=atoi(argv[1]);
    if(k <1){ //checks that k value is greater than 1
		printf("invalid number of Centers");
		return 1;
    }
    FILE *fp;
    fp=fopen(argv[2], "r");
    if(fp==NULL){ //chechs that the file opened properly
		perror("Failed to open file:");
		return 1;
    }
    day * data;
    data=(day*)malloc(sizeof(struct day));
    while((fgetc(fp))!='\n'){}//getting rid of the title line of the file
    
    int numDays=0;
    int high=-1;
    int low;
    int date;
    int month;
    int year;
    char  station[15];
    while(fscanf(fp,"%[^,],%d/%d/%d,%d,%d",station,&month, &date, &year, &high, &low)==6){//populates data from file
		numDays++;
		data=(day*)realloc(data, sizeof(struct day) * numDays);
		data[numDays-1].date=date;
		data[numDays-1].high=high;
		data[numDays-1].low=low;
		data[numDays-1].month=month;
		data[numDays-1].year=year;
		data[numDays-1].cluster=-1;
    }
    fclose(fp);
    //declares data for device
    day * d_data;
    hipMalloc((void **)&d_data, sizeof(struct day)*numDays);
    hipMemcpy(d_data, data, sizeof(struct day)*numDays, hipMemcpyHostToDevice);
    
    //create centers 
    center * centers;
    centers=(center*)malloc(sizeof(struct center)* k);
    for(int i=0; i<k; i++){//initilize centers to random data points
	centers[i].x=data[numDays/(i+2)].high;
	centers[i].y=data[numDays/(i+2)].low;

    } 
    
    //create centers for device
    center * d_centers;
    hipMalloc((void **)&d_centers, sizeof(struct center) *k);
    hipMemcpy(d_centers, centers, sizeof(struct center) *k, hipMemcpyHostToDevice);
    
    int temp = 1093; //random number non zero number
    int * s=&temp;

    int * d_s;//variable to count how many data points change clusters between iterations
    hipMalloc((void **)&d_s, sizeof(int));
    
    
    while(*s>0 ){
	*s=0;//reset s value
	int numB=numDays/512;
	cluster<<<numB, 512>>>(d_data, d_centers, k, numDays, d_s);//cluster data
	hipMemcpy(s, d_s, sizeof(int), hipMemcpyDeviceToHost);//retrieve d_S value from device
	if(*s>0){//compute new centers if any clusters changed
		int numT=((numDays/k)/32)*32; //assigns highest 
 		numT>512 ? numT=512 : numT=numT;
		setCenters<<<k, numT>>>(d_data, d_centers, k, numDays);
		/*cudaMemcpy(centers, d_centers, sizeof(struct center)*k, cudaMemcpyDeviceToHost);
		for( int h=0; h<k; h++){
			printf("x=%f y=%f\n", centers[h].x, centers[h].y);	
		}*/
	}
	
	
    }
    hipMemcpy(data, d_data, sizeof(struct day)*numDays, hipMemcpyDeviceToHost);
    fp=fopen("output.csv", "w");
    for(int i=0; i<k; i++){
	fprintf(fp, "Cluster %d,Center,x=%f, y=%f\nDate,High,Low\n", i+1, centers[i].x, centers[i].y);
	for(int j=0; j<numDays; j++){
		if(i==data[j].cluster){
			fprintf(fp, "%d/%d/%d,%f,%f,%d\n", data[j].month, data[j].date, data[j].year, data[j].high,data[j].low, data[j].cluster);
		}
	}
	fprintf(fp,"\n\n");
    }
    
       

    

    // Cleanup
   /* cudaFree(d_centers);
    cudaFree(d_s); 
    cudaFree(d_data);
    free(data); 
    free(s);
    free(centers);*/
    return 0;
}
