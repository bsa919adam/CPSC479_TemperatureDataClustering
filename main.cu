#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

typedef struct day{
	int month;
	int date;
	int year;
	double high;
	double low;
	int cluster=-1;
	int previous=-1;
}day;

typedef struct center{
	int x;
	int y;
}center;


__global__ void setCenters(int* a) {
	
	int N=0;
	for( int i=0; i< N; i++){
		a[blockIdx.x * N + threadIdx.x] += a[blockIdx.x * N + i] * a[i * N + threadIdx.x];
	}
	
}

__global__ void kluster(day* data, day* centers, day ** clusters){


}

int main(int  argc, char *argv[]) {
    printf("begin checks\n");
    if(argc < 3){ //checcks for proper number of Args
		printf("Missing Arguments");
		return 1;
    }
    int k=atoi(argv[1]);
    if(k <1){ //checks that k value is greater than 1
		printf("invalid number of Centers");
		return 1;
    }
    FILE *fp;
    fp=fopen(argv[2], "r");
    if(fp==NULL){ //chechs that the file opened properly
		perror("Failed to open file:");
		return 1;
    }
    day * data;
    char c=' ';
    data=(day*)malloc(sizeof(struct day));
    while((fgetc(fp))!='\n'){}//getting rid of the title line of the file
    
    int numDays=0;
    int high=-1;
    int low;
    int date;
    int month;
    int year;
    char  station[15];
    while(fscanf(fp,"%[^,],%d/%d/%d,%d,%d",station,&month, &date, &year, &high, &low)==6){//populates data from file
		numDays++;
		data=(day*)realloc(data, sizeof(struct day) * numDays);
		data[numDays-1].date=date;
		data[numDays-1].high=high;
		data[numDays-1].low=low;
		data[numDays-1].month=month;
		data[numDays-1].year=year;
    }
    day * d_data;
    //declares data for device
    hipMalloc((void **)&d_data, sizeof(struct day)*numDays);
    hipMemcpy(d_data, data, sizeof(struct day)*numDays);
  
    

    
       
 // hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    

    // Cleanup
     
    hipFree(d_data);
    free(data); 
    return 0;
}
