
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
//use high as x value low as y for clustering
typedef struct day{
	int month;
	int date;
	int year;
	double high;
	double low;
	int cluster;
}day;

typedef struct center{
	double x;
	double y;
}center;


__global__ void setCenters(day* data, center* centers, int k, int numDays) {
	__shared__ int nums[3];

	int * avgx =nums;
	int * avgy =&nums[1];
	int * n=&nums[2];
	*avgx=0;
	*avgy=0;
	*n=0;
	int i =0;
	int index;	
	while((index=threadIdx.x + blockDim.x*i) < numDays){
		if(data[index].cluster==blockIdx.x){
			atomicAdd(n, 1);
			atomicAdd(avgx, (int)data[index].high);
			atomicAdd(avgy, (int)data[index].low);
			
		}
		
		i++;
	}
	__syncthreads();
	if(threadIdx.x ==0){
		centers[blockIdx.x].x=(double)(*avgx)/(double)(*n);
		centers[blockIdx.x].y=(double)(*avgy)/(double)(*n);
	}
	
}

__global__ void cluster(day* data, center* centers, int k, int numDays, int * s){
	int numT=gridDim.x*blockDim.x;
	int i=0;
	int index;
	int cluster;
	*s=0;
	while((index=threadIdx.x +blockIdx.x * blockDim.x+ numT*i) < numDays){
		double min=1000;
		for( int j=0; j< k; j++){
			double x=data[index].high-centers[j].x;
			x=x*x;
			double y=data[index].low-centers[j].y;
			y=y*y;
			double dist=sqrt(x+y);
			if(dist< min){
				min=dist;
				cluster=j;
				
			}		
		}
		if(data[index].cluster!=cluster){
			atomicAdd(s, 1);
			data[index].cluster=cluster;
		}
		i++;
	} 
}
__global__ void processData(day* data, int * month_data, int k, int numDays){
	int i=0;
	int month_index = blockIdx.x * 12 + threadIdx.x; //index for month_data
	int data_index;//index for data
	int month = threadIdx.x + 1;//month to look for
	int cluster = blockIdx.x;//cluster to look for
	
	while((data_index = threadIdx.y + blockDim.y * i++) < numDays){
	
		if((data[data_index].cluster == cluster) && (data[data_index].month == month) ){
			atomicAdd(&month_data[month_index], 1);
		}
		
	}
}
int main(int  argc, char *argv[]) {
    printf("begin checks\n");
    if(argc < 3){ //checcks for proper number of Args
			printf("Missing Arguments");
			return 1;
    }
    int k=atoi(argv[1]);
    if(k <1){ //checks that k value is greater than 1
			printf("invalid number of Clusters");
			return 1;
    }
    FILE *fp;
    fp=fopen(argv[2], "r");
    if(fp==NULL){ //chechs that the file opened properly
			perror("Failed to open file:");
			return 1;
    }
    day * data;
    data=(day*)malloc(sizeof(struct day));
    while((fgetc(fp))!='\n'){}//getting rid of the title line of the file
    
    int numDays=0;
    int high=-1;
    int low;
    int date;
    int month;
    int year;
    char  station[15];
    while(fscanf(fp,"%[^,],%d/%d/%d,%d,%d",station,&month, &date, &year, &high, &low)==6){//populates data from file
			numDays++;
			data=(day*)realloc(data, sizeof(struct day) * numDays);
			data[numDays-1].date=date;
			data[numDays-1].high=high;
			data[numDays-1].low=low;
			data[numDays-1].month=month;
			data[numDays-1].year=year;
			data[numDays-1].cluster=-1;
		}
		
		fclose(fp);//close file
		
    //declares data for device
    day * d_data;
    hipMalloc((void **)&d_data, sizeof(struct day)*numDays);
    hipMemcpy(d_data, data, sizeof(struct day)*numDays, hipMemcpyHostToDevice);
    
    //create centers 
    center * centers;
    centers=(center*)malloc(sizeof(struct center)* k);
    for(int i=0; i<k; i++){//initilize centers to random data points
			centers[i].x=data[numDays/(i+2)].high;
			centers[i].y=data[numDays/(i+2)].low;

    } 
    
    //create centers for device
    center * d_centers;
    hipMalloc((void **)&d_centers, sizeof(struct center) *k);
    hipMemcpy(d_centers, centers, sizeof(struct center) *k, hipMemcpyHostToDevice);
    
    int temp = 1093; //random number non zero number
    int * s=&temp;

    int * d_s;//variable to count how many data points change clusters between iterations
    hipMalloc((void **)&d_s, sizeof(int));
    
    
    while(*s>0 ){
	  	*s=0;//reset s value
			int numB=numDays/512;

			cluster<<<numB, 512>>>(d_data, d_centers, k, numDays, d_s);//cluster data
		
			hipMemcpy(s, d_s, sizeof(int), hipMemcpyDeviceToHost);//retrieve d_S value from device
			
			if(*s>0){//compute new centers if any clusters changed
				int numT=((numDays/k)/32)*32; //assigns highest 
				numT>512 ? numT=512 : numT=numT;//checks that numt doesn't exceed 512
				
				setCenters<<<k, numT>>>(d_data, d_centers, k, numDays);
			
				/*cudaMemcpy(centers, d_centers, sizeof(struct center)*k, cudaMemcpyDeviceToHost);
				for( int h=0; h<k; h++){
					printf("x=%f y=%f\n", centers[h].x, centers[h].y);	
				}*/
			}	
		}
		//copy data back to device for printing
    hipMemcpy(data, d_data, sizeof(struct day)*numDays, hipMemcpyDeviceToHost);
		//open file for output
		fp=fopen("output.csv", "w");
		
		//print data to output in csv format
		for(int i=0; i<k; i++){
			fprintf(fp, "Cluster %d,Center,x=%f, y=%f\nDate,High,Low\n", i+1, centers[i].x, centers[i].y);
			for(int j=0; j<numDays; j++){
				if(i==data[j].cluster){
					fprintf(fp, "%d/%d/%d,%f,%f,%d\n", data[j].month, data[j].date, data[j].year, data[j].high,data[j].low, data[j].cluster);
				}
			}
			fprintf(fp,"\n\n");
    }
		//pointer to single dimensional array that is to 
		//hold summary of many days of each month are in each
		int * month_data;
		month_data=(int*)malloc(k*12*sizeof(int));
		int * d_month_data;//device copy
		hipMalloc((void **)&d_month_data, k*12*sizeof(int) );
		
		dim3 threads(12, 32);
		processData<<<k, threads>>>(d_data, d_month_data, k, numDays );
		hipMemcpy(month_data, d_month_data, k * 12 *sizeof(int), hipMemcpyDeviceToHost);
		fprintf(fp, "Cluster,JAN,FEB,MAR,APR,MAY,JUN,JUL,AUG,SEP,OCT,NOV,DEC\n");
		printf("%6s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s%5s\n","Cluster","JAN", "FEB", "MAR","APR","MAY","JUN","JUL","AUG","SEP","OCT","NOV","DEC");
		for( int i = 0; i< k; i++){
			fprintf(fp, "%d", i+1);
			printf("%6d", i+1);
			for( int j = 0; j < 12; j++){
				fprintf(fp, ",%d",month_data[i * 12 + j]);
				printf("%5d",month_data[i * 12 + j]);
			}
			fprintf(fp, "\n");
			printf("\n");
		}

    // Cleanup
    hipFree(d_centers);
    hipFree(d_s); 
		hipFree(d_data);
		hipFree(d_month_data);
    free(data); 
    // free(s);
		free(centers);
		free(month_data);
    return 0;
}
