
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 32
__global__ void square(int* a) {
	
	
	for( int i=0; i< N; i++){
		a[blockIdx.x * N + threadIdx.x] += a[blockIdx.x * N + i] * a[i * N + threadIdx.x];
	}
	
}

void random_ints(int* arr, int row, int colum) {
    for (int a = 0; a < row; a++) {
	for( int b=0; b<colum; b++){
        	arr[a * N + b] = rand() % 100 + 1;
	}
    }
}

int main(void) {
    int *a;        // host copy of a
    int *d_a;    // device copy of a
    int size = N * N * sizeof(int);
    srand(time(NULL));
    printf("line 29 size=%d\n",size);
    // Alloc space for host copy of a and setup input values
    a = (int *)malloc(size); 
    random_ints(a, N, N);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N blocks
    square<<<N,N>>>(d_a);
    // Copy result back to host
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
	for(int j = 0; j< N; j++){
        	printf("a[%d][%d] = %d\n", i,j, a[i*N+j]);
	}
    }

    // Cleanup
    free(a); 
    hipFree(d_a); 
    return 0;
    }
