
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

typedef struct day{
	int month;
	int date;
	int year;
	double high;
	double low;
}day;



__global__ void setCenters(int* a) {
	
	int N=0;
	for( int i=0; i< N; i++){
		a[blockIdx.x * N + threadIdx.x] += a[blockIdx.x * N + i] * a[i * N + threadIdx.x];
	}
	
}

__global__ void kluster(day* data, day* centers, day ** clusters){


}

int main(int  argc, char *argv[]) {
    printf("begin checks\n");
    if(argc < 3){
	printf("Missing Arguments");
	return 1;
    }
    printf("check2\n");
    int k=atoi(argv[1]);
    if(k <1){
	printf("invalid number of Centers");
	return 1;
    }
    printf("check3\n");
    FILE *fp;
    fp=fopen(argv[2], "r");
    if(fp==NULL){
	perror("Failed to open file:");
	return 1;
    }
    printf("checks done\n");
    day * data;
    char c=' ';
    data=(day*)malloc(sizeof(struct day));
    printf("starting line skip\n");
    while((c=fgetc(fp))!='\n'){//getting rid of the title line of the file
	//printf("%c", c);

    }
    printf("line skipped\n");
    int numDays=0;
    int high=-1;
    int low;
    int date;
    int month;
    int year;
    char  station[15];
    while(fscanf(fp,"%[^,],%d/%d/%d,%d,%d",station,&month, &date, &year, &high, &low)==6){
	numDays++;
	data=(day*)realloc(data, sizeof(struct day) * numDays);
	data[numDays-1].date=date;
	data[numDays-1].high=high;
	data[numDays-1].low=low;
	data[numDays-1].month=month;
	data[numDays-1].year=year;
	printf("%d\n", numDays);
    }
    printf("%e\n", data[0].high);
    printf("%e\n", data[numDays-1].high);
    printf("%d\n", numDays);

    
 //   cudaMalloc((void **)&d_a, size);
    
   // cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);

    
       
 // cudaMemcpy(a, d_a, size, cudaMemcpyDeviceToHost);
    

    // Cleanup
     
   // cudaFree(d_a);
    free(data); 
    return 0;
}
